#include "hip/hip_runtime.h"
/**
 * \file dnn/src/cuda/resize/backward.cu
 * MegEngine is Licensed under the Apache License, Version 2.0 (the "License")
 *
 * Copyright (c) 2014-2020 Megvii Inc. All rights reserved.
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT ARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 */
#include "src/cuda/resize/common.cuh"
#include "src/cuda/resize/common.h"

#include "src/cuda/utils.cuh"

namespace megdnn {
namespace cuda {
namespace resize {

__global__ void resize_bwd_kernel(const float* hidden, float* dst, int N, int C,
                                  int IH, int IW, int OH, int OW, float scale_h,
                                  float scale_w) {
    int n = blockIdx.z;
    int ow = blockIdx.x * blockDim.x + threadIdx.x;
    int oh = blockIdx.y * blockDim.y + threadIdx.y;
    hidden += n * C * OH * OW;
    dst += n * C * IH * IW;
    if (ow < OW && oh < OH) {
        float alphah, alphaw;
        int ih0, iw0;
        get_origin_coord(scale_h, IH, oh, alphah, ih0);
        get_origin_coord(scale_w, IW, ow, alphaw, iw0);

        int ih1 = ih0 + 1;
        int iw1 = iw0 + 1;

        float nalphaw = 1.0f - alphaw;
        float nalphah = 1.0f - alphah;
        for (int c = 0; c < C; ++c) {
            atomicAdd(dst + ih0 * IW + iw0,
                      hidden[oh * OW + ow] * nalphaw * nalphah);
            atomicAdd(dst + ih0 * IW + iw1,
                      hidden[oh * OW + ow] * alphaw * nalphah);
            atomicAdd(dst + ih1 * IW + iw0,
                      hidden[oh * OW + ow] * nalphaw * alphah);
            atomicAdd(dst + ih1 * IW + iw1,
                      hidden[oh * OW + ow] * alphaw * alphah);
            hidden += OH * OW;
            dst += IH * IW;
        }
    }
}

void backward_data_proxy(const float* diff, float* grad, int N, int C, int IH,
                         int IW, int OH, int OW, hipStream_t stream) {
    const int BY = 16, BX = 32;
    {
        dim3 threads(BX, BY);
        dim3 blocks((OW + BX - 1) / BX, (OH + BY - 1) / BY, N);
        cuda_check(hipMemsetAsync(grad, 0, sizeof(float) * N * C * IH * IW,
                                   stream));
        float scale_h = static_cast<float>(OH) / IH;
        float scale_w = static_cast<float>(OW) / IW;
        resize_bwd_kernel<<<blocks, threads, 0, stream>>>(
                diff, grad, N, C, IH, IW, OH, OW, scale_h, scale_w);
    }
    after_kernel_launch();
}

}  // namespace resize
}  // namespace cuda
}  // namespace megdnn

// vim: syntax=cpp.doxygen
