/**
 * \file dnn/src/cuda/relayout/kern.cu
 * MegEngine is Licensed under the Apache License, Version 2.0 (the "License")
 *
 * Copyright (c) 2014-2020 Megvii Inc. All rights reserved.
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT ARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 */

#include "megdnn/basic_types.h"
#include "src/cuda/elemwise_helper.cuh"
#include "src/cuda/utils.cuh"
#include "src/cuda/relayout/kern.cuh"

namespace megdnn {
namespace cuda {

void copy_noncontig_general(const TensorND &dst, const TensorND &src, hipStream_t stream) {
    ElemwiseOpParamN<2> param;
    param[0] = dst;
    param[1] = src;

#define RUN(_dt)                                                        \
    do {                                                                \
        typedef DTypeTrait<dtype::_dt>::ctype ctype;                    \
        param[0].layout.dtype = param[1].layout.dtype = dtype::_dt();   \
        param.init_from_given_tensor();                                 \
        param.assert_initialized();                                     \
        noncontig_general_intl::UserOpInvoker<ctype, 2>(param, stream); \
        return;                                                         \
    } while (0)

    switch (dst.layout.dtype.size()) {
        case 1:
            RUN(Byte);
        case 2:
            RUN(Float16);
        case 4:
            RUN(Int32);
    }
    megdnn_assert(0, "bad dtype size");
}

} // namespace cuda
} // namespace megdnn

// vim: ft=cpp syntax=cpp.doxygen
